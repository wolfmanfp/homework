/**
 * The program must read in an NxN matrix with floating point numbers. The program must determine
 * the index of columns, which contains one element that is equal to the average of the values in
 * the same column. The outputs are the indices of the columns.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



double* readMatrix(int size, char *filename) {
	double *vector;

  FILE *fp = fopen(filename, "r");
	if (fp == NULL) {
		printf("A fajl nem talalhato!");
		exit(1);
	}

	vector = (double *) malloc(size * size * sizeof(double));
  int i = 0;
	while (fscanf(fp, "%lf ", &vector[i]) != EOF) {
		i++;
	}

	fclose(fp);
	return vector;
}

__global__ void findIndicesKernel(int size, double *vector, int *indices) {
  for (int col = 0; col < size; col++) {
    double sum = 0.0;
    for (int row = 0; row < size; row++) {
      sum += vector[col + row * size];
    }
    double avg = sum / size;

    for (int row = 0; row < size; row++) {
      if (vector[col + row * size] == avg) {
        indices[col] = col;
        break;
      }
    }
  }
}

void printMeasuredTime(int size, double time) {
  FILE *fp = fopen("time.txt", "w");
  fprintf(fp, "%dx%d matrix: %.8lf s", size, size, time);
  fclose(fp);
}

int* findIndices(int size, double *vector) {
  int *indices, *device_indices;
  double *device_vector;

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  indices = (int *)malloc(size * sizeof(int));
  for (int i = 0; i < size; i++)
  {
    indices[i] = -1;
  }

  hipMalloc((void **)&device_vector, size * size * sizeof(double));
  hipMalloc((void **)&device_indices, size * sizeof(int));
  hipMemcpy(device_vector, vector, size * size, hipMemcpyHostToDevice);
  hipMemcpy(device_indices, indices, size, hipMemcpyHostToDevice);

  hipEventRecord(start);
  findIndicesKernel<<<1, 1>>>(size, device_vector, device_indices);
  hipEventRecord(end);

  hipMemcpy(indices, device_indices, size, hipMemcpyDeviceToHost);
  hipFree(device_vector);
  hipFree(device_indices);

  hipEventSynchronize(end);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, end);
  printMeasuredTime(size, milliseconds / 1000);

  return indices;
}

void printResults(int size, int *indices) {
	FILE *fp = fopen("output.txt", "w");
  for (int i = 0; i < size; i++) {
    if (indices[i] != -1) {
      fprintf(fp, "%d ", indices[i]);
    }
  }
	fclose(fp);
}

int main(int argc, char **argv) {
  if (argc < 2) {
    printf("Kerem, adja meg a matrix meretet\n es az azt tartalmazo fajl\neleresi utvonalat!\nPelda: hf1 5 matrix.txt");
    return 1;
  }

  int size = strtod(argv[1], NULL);
  double *vector = readMatrix(size, argv[2]);
  int *indices = findIndices(size, vector);
  printResults(size, indices);

  free(vector);
  free(indices);
  return 0;
}