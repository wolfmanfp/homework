/**
 * The program must read in an NxN matrix with floating point numbers. The program must determine
 * the index of columns, which contains one element that is equal to the average of the values in
 * the same column. The outputs are the indices of the columns.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



double* readMatrix(int size, char *filename) {
	double *vector;

  FILE *fp = fopen(filename, "r");
	if (fp == NULL) {
		printf("A fajl nem talalhato!");
		exit(1);
	}

	vector = (double *) malloc(size * size * sizeof(double));
  int i = 0;
	while (fscanf(fp, "%lf ", &vector[i]) != EOF) {
		i++;
	}

	fclose(fp);
	return vector;
}

__global__ void findIndicesKernel(int size, double *vector, int *indices) {
  for (int col = 0; col < size; col++) {
    double sum = 0.0;
    for (int row = 0; row < size; row++) {
      sum += vector[col + row * size];
    }
    double avg = sum / size;

    indices[col] = -1;
    for (int row = 0; row < size; row++) {
      if (vector[col + row * size] == avg) {
        indices[col] = col;
        break;
      }
    }
  }
}

void printMeasuredTime(int size, double time) {
  FILE *fp = fopen("time.txt", "w");
  fprintf(fp, "%dx%d matrix: %.8lf s", size, size, time);
  fclose(fp);
}

int* findIndices(int size, double *vector) {
  int *indices, *device_indices;
  double *device_vector;
  size_t vector_size = size * size * sizeof(double);
  size_t indices_size = size * sizeof(int);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  indices = (int *)malloc(size * sizeof(int));
  hipMalloc((void **)&device_vector, vector_size);
  hipMalloc((void **)&device_indices, indices_size);
  hipMemcpy(device_vector, vector, vector_size, hipMemcpyHostToDevice);

  hipEventRecord(start);
  findIndicesKernel<<<1, 1>>>(size, device_vector, device_indices);
  hipEventRecord(end);

  hipMemcpy(indices, device_indices, indices_size, hipMemcpyDeviceToHost);
  hipFree(device_vector);
  hipFree(device_indices);

  hipEventSynchronize(end);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, end);
  printMeasuredTime(size, milliseconds / 1000);

  return indices;
}

void printResults(int size, int *indices) {
	FILE *fp = fopen("output.txt", "w");
  for (int i = 0; i < size; i++) {
    if (indices[i] != -1) {
      fprintf(fp, "%d ", indices[i]);
    }
  }
	fclose(fp);
}

int main(int argc, char **argv) {
  if (argc < 2) {
    printf("Kerem, adja meg a matrix meretet\n es az azt tartalmazo fajl\neleresi utvonalat!\nPelda: hf1 5 matrix.txt");
    return 1;
  }

  int size = strtod(argv[1], NULL);
  double *vector = readMatrix(size, argv[2]);
  int *indices = findIndices(size, vector);
  printResults(size, indices);

  free(vector);
  free(indices);
  return 0;
}